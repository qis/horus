#include "hip/hip_runtime.h"
#include "eye.hpp"
#include "mulxp_hash.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <opencv2/core/cuda/common.hpp>
#include <algorithm>
#include <numeric>
#include <cassert>

#define SHAPES_FILTER_FAST 0

namespace horus {
namespace {

using cv::cuda::device::divUp;

// Sets mask values to:
// - 0x02 if they have too many neighbors.
__global__ void mask_filter(uchar* data, size_t step)
{
  constexpr auto r = 3;
  constexpr auto d = r * 2 + 1;
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < r || x >= eye::vw - r || y < r || y >= eye::vh - r) {
    return;
  }
  const auto di = data + y * step + x;
  const auto dv = *di;
  if (!dv) {
    return;
  }
  auto neighbors = -1;
  auto si = di - step * r - r;
  for (auto i = 0; i < d; i++) {
    for (auto j = 0; j < d; j++) {
      const auto sc = *si++;
      if (sc && ++neighbors > 9) {
        *di = 0x02;
        return;
      }
    }
    si += step - d;
  }
}

// Sets mask values to:
// - 0x00 if they were outside the filter radius.
__global__ void mask_shrink(uchar* data, size_t step)
{
  constexpr auto r = 3;
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= eye::vw || y >= eye::vh) {
    return;
  }
  const auto di = data + y * step + x;
  if (x < r || x >= eye::vw - r || y < r || y >= eye::vh - r) {
    *di = 0x00;
  }
}

// Sets mask values to:
// - 0x04 if they have a 0x02 neighbor.
__global__ void mask_dilate(uchar* data, size_t step)
{
  constexpr auto r = 3;
  constexpr auto d = r * 2 + 1;
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < r || x >= eye::vw - r || y < r || y >= eye::vh - r) {
    return;
  }
  const auto di = data + y * step + x;
  const auto dv = *di;
  if (dv != 0x01) {
    return;
  }
  auto si = di - step * r - r;
  for (auto i = 0; i < d; i++) {
    for (auto j = 0; j < d; j++) {
      const auto sc = *si++;
      if (sc == 0x02) {
        *di = 0x04;
        return;
      }
    }
    si += step - d;
  }
}

// Sets 0x01 bit on filtered or dilated mask values that have a neighbor with the 0x01 bit set.
__global__ void mask_erode(uchar* data, size_t step)
{
  constexpr auto r = 1;
  constexpr auto d = r * 2 + 1;
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < r || x >= eye::vw - r || y < r || y >= eye::vh - r) {
    return;
  }
  const auto di = data + y * step + x;
  const auto dv = *di;
  if (dv != 0x02 && dv != 0x04) {
    return;
  }
  auto si = di - step * r - r;
  for (auto i = 0; i < d; i++) {
    for (auto j = 0; j < d; j++) {
      const auto sc = *si++;
      if (sc & 0x01) {
        *di |= 0x01;
        return;
      }
    }
    si += step - d;
  }
}

// Draws mask values as 0x01 if the 0x01 bit is set.
__global__ void mask_scan(const uchar* data, size_t data_step, uchar* scan, size_t scan_step)
{
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= eye::vw || y >= eye::vh) {
    return;
  }
  scan[y * scan_step + x] = data[y * data_step + x] & 0x01 ? 0x01 : 0x00;
}

// Sets RGBA pixel value.
__device__ __forceinline__ void device_set(uchar* di, std::uint32_t color) noexcept
{
  *di++ = static_cast<uchar>(color >> 24 & 0xFF);
  *di++ = static_cast<uchar>(color >> 16 & 0xFF);
  *di++ = static_cast<uchar>(color >> 8 & 0xFF);
  *di = static_cast<uchar>(color & 0xFF);
}

// Draws mask values as separate colors.
__global__ void mask_draw(const uchar* data, size_t data_step, uchar* view, size_t view_step)
{
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= eye::vw || y >= eye::vh) {
    return;
  }
  // clang-format off
  auto di = view + y * view_step + x * 4;
  switch (data[y * data_step + x]) {
  case 0x00: device_set(di, 0x00000000); break;  // None   |       | Transparent
  case 0x01: device_set(di, 0x64DD17FF); break;  // Mask   |       | A700 Light Green
  case 0x02: device_set(di, 0xD50000FF); break;  // Filter |       | A700 Red
  case 0x03: device_set(di, 0xEEFF41FF); break;  // Filter | Erode | A200 Lime
  case 0x04: device_set(di, 0xFF9100FF); break;  // Dilate |       | A400 Orange
  case 0x05: device_set(di, 0xF4FF81FF); break;  // Dilate | Erode | A100 Lime
  default:   device_set(di, 0xE040FBFF); break;  // Error  |       | A200 Purple
  }
  // clang-format on
}

// Sets shapes values to:
// - 0x00 if they are border pixels.
// - 0x02 if they are near a 0x01 value.
__global__ void shapes_dilate(uchar* data, size_t step)
{
  constexpr auto r = 2;
  constexpr auto d = r * 2 + 1;
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= eye::vw || y >= eye::vh) {
    return;
  }
  const auto di = data + y * step + x;
  if (x < r || x >= eye::vw - r || y < r || y >= eye::vh - r) {
    *di = 0x00;
    return;
  }
  if (*di) {
    return;
  }
  auto si = di - r * step - r;
  for (auto i = 0; i < d; i++) {
    for (auto j = 0; j < d; j++) {
      if (*si++ == 0x01) {
        *di = 0x02;
        return;
      }
    }
    si += step - d;
  }
}

// Sets shapes values to:
// - 0x01 if they are not masked.
// - 0x05 if they are masked.
__global__ void shapes_mask(uchar* data, size_t step, const uchar* mask, size_t mask_step)
{
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= eye::vw || y >= eye::vh) {
    return;
  }
  const auto di = data + y * step + x;
  if (!*di) {
    return;
  }
  *di = mask[y * mask_step + x] ? 0x05 : 0x01;
}

// Sets shapes values to:
// - 0x02 if they are 0x01 and should be filtered.
// - 0x03 if they are 0x01 and are a filtered border.
#if SHAPES_FILTER_FAST
__global__ void shapes_filter(uchar* data, size_t step)
{
  constexpr auto r = 1;
  constexpr auto d = r * 2 + 1;
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < r || x >= eye::vw - r || y < r || y >= eye::vh - r) {
    return;
  }
  const auto di = data + y * step + x;
  if (*di != 0x01) {
    return;
  }
  auto filter = false;
  auto border = false;
  auto si = di - r * step - r;
  for (auto i = 0; i < d; i++) {
    for (auto j = 0; j < d; j++) {
      const auto sc = *si++;
      filter |= !sc || (sc & 0x02) != 0x00;
      border |= sc > 0x02;
    }
    si += step - d;
  }
  if (filter) {
    *di = border ? 0x03 : 0x02;
  }
}
#else
__global__ void shapes_filter(uchar* data, size_t step)
{
  constexpr auto r = 1;
  constexpr auto d = r * 2 + 1;
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < r || x >= eye::vw - r || y < r || y >= eye::vh - r) {
    return;
  }
  const auto di = data + y * step + x;
  if (*di != 0x01) {
    return;
  }
  auto filter = 0;
  auto border = 0;
  auto si = di - r * step - r;
  const auto sv = *si;
  for (auto i = 0; i < d; i++) {
    for (auto j = 0; j < d; j++) {
      const auto sc = *si++;
      if (filter < 2) {
        if (!sc || (sc & 0x02) != 0x00) {
          ++filter;
        } else {
          filter = 0;
        }
      }
      if (sc > 0x02) {
        ++border;
      }
    }
    si += step - d;
  }
  if (filter > 1 || (filter == 1 && (!sv || (sv & 0x02) != 0x00))) {
    *di = border > 1 ? 0x03 : 0x02;
  }
}
#endif

// Sets shapes values to:
// - 0x03 if they are 0x01 and have a 0x03 neighbor.
__global__ void shapes_close(uchar* data, size_t step)
{
  constexpr auto r = 1;
  constexpr auto d = r * 2 + 1;
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < r || x >= eye::vw - r || y < r || y >= eye::vh - r) {
    return;
  }
  const auto di = data + y * step + x;
  if (*di != 0x01) {
    return;
  }
  auto si = di - r * step - r;
  for (auto i = 0; i < d; i++) {
    for (auto j = 0; j < d; j++) {
      if (*si++ == 0x03) {
        *di = 0x03;
        return;
      }
    }
    si += step - d;
  }
}

// Sets shapes values to:
// - 0x02 if they are 0x03 and have a 0x02 neighbor.
__global__ void shapes_erode(uchar* data, size_t step)
{
  constexpr auto r = 1;
  constexpr auto d = r * 2 + 1;
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < r || x >= eye::vw - r || y < r || y >= eye::vh - r) {
    return;
  }
  const auto di = data + y * step + x;
  if (*di != 0x03) {
    return;
  }
  auto si = di - r * step - r;
  for (auto i = 0; i < d; i++) {
    for (auto j = 0; j < d; j++) {
      if (*si++ == 0x02) {
        *di = 0x02;
        return;
      }
    }
    si += step - d;
  }
}

// Draws shapes values as 0x01 if the 0x01 bit is set.
__global__ void shapes_scan(const uchar* data, size_t data_step, uchar* scan, size_t scan_step)
{
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= eye::vw || y >= eye::vh) {
    return;
  }
  scan[y * scan_step + x] = data[y * data_step + x] & 0x01 ? 0x01 : 0x00;
}

__global__ void shapes_draw(const uchar* data, size_t data_step, uchar* view, size_t view_step)
{
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= eye::vw || y >= eye::vh) {
    return;
  }
  // clang-format off
  auto di = view + y * view_step + x * 4;
  switch (data[y * data_step + x]) {
  case 0x00: device_set(di, 0x00000000); break;  // None     | Transparent
  case 0x01: device_set(di, 0x689F387F); break;  // Shape    | 700 Light Green
  case 0x02: device_set(di, 0xD32F2F7F); break;  // Filtered | 700 Red
  case 0x03: device_set(di, 0xFFD6007F); break;  // Border   | A700 Yellow
  case 0x05: device_set(di, 0x64DD17FF); break;  // Outline  | A700 Light Green
  default:   device_set(di, 0xE040FBFF); break;  // Error    | A200 Purple
  }
  // clang-format on
}

}  // namespace

const cv::Point eye::vc{ vw / 2, vh / 2 };

eye::eye()
{
  scan_.setTo(cv::Scalar(0));
  scan_hash_ = mulxp3_hash(scan_.data, scan_.step * scan_.rows, 0);
  freetype_ = cv::freetype::createFreeType2();
  freetype_->loadFontData("C:/OBS/horus/res/fonts/PixelOperatorMono.ttf", 0);
}

bool eye::scan(const cv::Mat& scan) noexcept
{
  // Verify scan type and size.
  assert(scan.type() == CV_8UC1);
  assert(scan.cols == sw);
  assert(scan.rows == sh);

  // Measure scan duration.
  const auto tp0 = clock::now();

  // Resize scan (120 μs).
  cv::resize(scan, scan_, { vw, vh }, 1.0 / vf, 1.0 / vf, cv::INTER_AREA);

  // Update hash (30 μs).
  const auto scan_hash = mulxp3_hash(scan_.data, scan_.step * scan_.rows, 0);
  if (scan_hash == scan_hash_) {
    return false;
  }
  scan_duration_ = clock::now() - tp0;
  scan_hash_ = scan_hash;

  hulls_ready_ = false;
  polygons_ready_ = false;
  return true;
}

const std::vector<eye::polygon>& eye::hulls() noexcept
{
  if (hulls_ready_) {
    return hulls_;
  }

  const auto tp0 = clock::now();
  mask_data_.upload(scan_);
#ifndef __INTELLISENSE__
  const dim3 block(16, 16);
  const dim3 grid(divUp(eye::vw, block.x), divUp(eye::vh, block.y));
  mask_filter<<<grid, block>>>(mask_data_.data, mask_data_.step);
  assert(hipGetLastError() == hipSuccess);
  mask_shrink<<<grid, block>>>(mask_data_.data, mask_data_.step);
  assert(hipGetLastError() == hipSuccess);
  mask_dilate<<<grid, block>>>(mask_data_.data, mask_data_.step);
  assert(hipGetLastError() == hipSuccess);
  for (auto i = 0; i < 6; i++) {
    mask_erode<<<grid, block>>>(mask_data_.data, mask_data_.step);
    assert(hipGetLastError() == hipSuccess);
  }
  mask_scan<<<grid, block>>>(mask_data_.data, mask_data_.step, mask_view_.data, mask_view_.step);
  assert(hipGetLastError() == hipSuccess);
#endif
  mask_view_.download(mask_);

  const auto tp1 = clock::now();
  mask_duration_ = tp1 - tp0;

  cv::findContours(mask_, contours_, hierarchy_, cv::RETR_EXTERNAL, cv::CHAIN_APPROX_SIMPLE);

  const auto tp2 = clock::now();
  contours_duration_ = tp2 - tp1;

  hulls_.resize(contours_.size());
  for (std::size_t i = 0, size = contours_.size(); i < size; i++) {
    cv::convexHull(contours_[i], hulls_[i]);
  }

  const auto tp3 = clock::now();
  hulls_duration_ = tp3 - tp2;

  while (true) {
    const auto se = hulls_.end();
    for (auto si = hulls_.begin(); si != se; ++si) {
      const auto srect = cv::boundingRect(*si);
      if (srect.width < 3 || srect.height < 9) {
        hulls_.erase(si);
        goto repeat;
      }
      const auto sl = srect.x;
      const auto sr = srect.x + srect.width;
      const auto st = srect.y;
      const auto sb = srect.y + srect.height;
      for (auto di = std::next(si); di != se; ++di) {
        const auto drect = cv::boundingRect(*di);
        const auto dl = drect.x;
        const auto dr = drect.x + drect.width;
        const auto dt = drect.y;
        const auto db = drect.y + drect.height;
        if (sr < dl - 8 || sl > dr + 8 || sb < dt - 32 || st > db + 32) {
          continue;
        }
        si->reserve(si->size() + di->size());
        si->insert(si->end(), di->begin(), di->end());
        cv::convexHull(*si, *di);
        hulls_.erase(si);
        goto repeat;
      }
    }
    break;
  repeat:
    continue;
  }

  const auto tp4 = clock::now();
  groups_duration_ = tp4 - tp3;

  hulls_ready_ = true;
  return hulls_;
}

const std::vector<eye::polygon>& eye::polygons() noexcept
{
  if (polygons_ready_) {
    return polygons_;
  }
  if (!hulls_ready_) {
    hulls();
  }

  const auto tp0 = clock::now();
  shapes_.setTo(cv::Scalar(0));
  for (const auto& hull : hulls_) {
    const auto rect = cv::boundingRect(hull);
    cv::fillPoly(shapes_, hull, cv::Scalar(1), cv::LINE_4);
  }

  shapes_data_.upload(shapes_);
#ifndef __INTELLISENSE__
  const dim3 block(16, 16);
  const dim3 grid(divUp(eye::vw, block.x), divUp(eye::vh, block.y));
  shapes_dilate<<<grid, block>>>(shapes_data_.data, shapes_data_.step);
  assert(hipGetLastError() == hipSuccess);
  shapes_mask<<<grid, block>>>(shapes_data_.data, shapes_data_.step, mask_data_.data, mask_data_.step);
  assert(hipGetLastError() == hipSuccess);
  for (auto i = 0; i < 16; i++) {
    for (auto j = 0; j < 4; j++) {
      shapes_filter<<<grid, block>>>(shapes_data_.data, shapes_data_.step);
      assert(hipGetLastError() == hipSuccess);
    }
    shapes_close<<<grid, block>>>(shapes_data_.data, shapes_data_.step);
    assert(hipGetLastError() == hipSuccess);
  }
  shapes_erode<<<grid, block>>>(shapes_data_.data, shapes_data_.step);
  assert(hipGetLastError() == hipSuccess);
  shapes_scan<<<grid, block>>>(shapes_data_.data, shapes_data_.step, shapes_view_.data, shapes_view_.step);
  assert(hipGetLastError() == hipSuccess);
#endif
  shapes_view_.download(shapes_);

  const auto tp1 = clock::now();
  shapes_duration_ = tp1 - tp0;

  cv::findContours(shapes_, contours_, hierarchy_, cv::RETR_EXTERNAL, cv::CHAIN_APPROX_SIMPLE);
  polygons_.resize(contours_.size());
  for (std::size_t i = 0, size = contours_.size(); i < size; i++) {
    cv::approxPolyDP(contours_[i], polygons_[i], 2.0, true);
  }

  const auto tp2 = clock::now();
  polygons_duration_ = tp2 - tp1;

  polygons_ready_ = true;
  return polygons_;
}

clock::duration eye::draw_scan(cv::Mat& overlay) noexcept
{
  assert(overlay.type() == CV_8UC4);
  assert(overlay.cols == vw);
  assert(overlay.rows == vh);

  overlay.setTo(scalar(0x64DD17FF), scan_);  // A700 Light Green

  return scan_duration_;
}

clock::duration eye::draw_mask(cv::Mat& overlay) noexcept
{
  assert(overlay.type() == CV_8UC4);
  assert(overlay.cols == vw);
  assert(overlay.rows == vh);

  if (!hulls_ready_) {
    hulls();
  }

#ifndef __INTELLISENSE__
  const dim3 block(16, 16);
  const dim3 grid(divUp(sw, block.x), divUp(sh, block.y));
  mask_draw<<<grid, block>>>(mask_data_.data, mask_data_.step, view_.data, view_.step);
  assert(hipGetLastError() == hipSuccess);
#endif
  view_.download(overlay);

  return mask_duration_;
}

clock::duration eye::draw_contours(cv::Mat& overlay) noexcept
{
  assert(overlay.type() == CV_8UC4);
  assert(overlay.cols == vw);
  assert(overlay.rows == vh);

  if (!hulls_ready_) {
    hulls();
  }

  overlay.setTo(scalar(0x64DD17FF), mask_);  // A700 Light Green

  return contours_duration_;
}

clock::duration eye::draw_groups(cv::Mat& overlay) noexcept
{
  draw_contours(overlay);

  for (const auto& hull : hulls_) {
    cv::rectangle(overlay, cv::boundingRect(hull), scalar(0x00B0FFFF), 1, cv::LINE_8);  // A400 Light Blue
  }

  return groups_duration_;
}

clock::duration eye::draw_hulls(cv::Mat& overlay) noexcept
{
  assert(overlay.type() == CV_8UC4);
  assert(overlay.cols == vw);
  assert(overlay.rows == vh);

  if (!hulls_ready_) {
    hulls();
  }

  cv::fillPoly(overlay, hulls_, scalar(0x64DD1760), cv::LINE_4);            // A700 Light Green
  cv::polylines(overlay, hulls_, true, scalar(0x64DD17FF), 1, cv::LINE_4);  // A700 Light Green

  return hulls_duration_;
}

clock::duration eye::draw_shapes(cv::Mat& overlay) noexcept
{
  assert(overlay.type() == CV_8UC4);
  assert(overlay.cols == vw);
  assert(overlay.rows == vh);

  if (!polygons_ready_) {
    polygons();
  }

#ifndef __INTELLISENSE__
  const dim3 block(16, 16);
  const dim3 grid(divUp(sw, block.x), divUp(sh, block.y));
  shapes_draw<<<grid, block>>>(shapes_data_.data, shapes_data_.step, view_.data, view_.step);
  assert(hipGetLastError() == hipSuccess);
#endif
  view_.download(overlay);

  return shapes_duration_;
}

clock::duration eye::draw_polygons(cv::Mat& overlay) noexcept
{
  assert(overlay.type() == CV_8UC4);
  assert(overlay.cols == vw);
  assert(overlay.rows == vh);

  if (!polygons_ready_) {
    polygons();
  }

  cv::fillPoly(overlay, polygons_, scalar(0x64DD1760), cv::LINE_4);            // A700 Light Green
  cv::polylines(overlay, polygons_, true, scalar(0x64DD17FF), 1, cv::LINE_4);  // A700 Light Green

  return polygons_duration_;
}

void eye::draw(
  cv::Mat& overlay,
  cv::Point position,
  const std::string& text,
  int height,
  std::uint32_t fg,
  std::uint32_t bg) noexcept
{
  const auto fgc = scalar(fg);
  const auto bgc = scalar(bg);
  if (freetype_ && !freetype_->empty()) {
    if (bg & 0xFF) {
      freetype_->putText(overlay, text, position, height, bgc, 2, cv::LINE_AA, false);
    }
    freetype_->putText(overlay, text, position, height, fgc, 1, cv::LINE_4, false);
    return;
  }
  const auto font = cv::FONT_HERSHEY_PLAIN;
  const auto scale = height / 16.0;
  if (bg & 0xFF) {
    cv::putText(overlay, text, position, font, scale, bgc, 2, cv::LINE_AA);
  }
  cv::putText(overlay, text, position, font, scale, fgc, 1, cv::LINE_4);
}

void eye::draw(cv::Mat& overlay, cv::Point point, std::uint32_t fg, std::uint32_t bg) noexcept
{
  const auto set = [](uchar* di, std::uint32_t color, unsigned count) noexcept {
    for (unsigned i = 0; i < count; i++) {
      *di++ = static_cast<uchar>(color >> 24 & 0xFF);
      *di++ = static_cast<uchar>(color >> 16 & 0xFF);
      *di++ = static_cast<uchar>(color >> 8 & 0xFF);
      *di++ = static_cast<uchar>(color & 0xFF);
    }
    return di;
  };

  if (point.x < 2 || point.x > eye::sw - 3 || point.y < 2 || point.y > eye::sh - 3) {
    return;
  }

  const auto x = static_cast<long>(point.x);
  const auto y = static_cast<long>(point.y);

  const size_t step = overlay.step;

  auto di = overlay.data + (y - 2) * step + (x - 2) * 4;  // dst iterator

  // Line 1.
  di = set(di, bg, 4);
  di = di + step - 5 * 4;

  // Line 2.
  di = set(di, bg, 2);
  di = set(di, fg, 2);
  di = set(di, bg, 2);
  di = di + step - 6 * 4;

  // Line 3.
  di = set(di, bg, 1);
  di = set(di, fg, 4);
  di = set(di, bg, 1);
  di = di + step - 6 * 4;

  // Line 4.
  di = set(di, bg, 1);
  di = set(di, fg, 4);
  di = set(di, bg, 1);
  di = di + step - 6 * 4;

  // Line 5.
  di = set(di, bg, 2);
  di = set(di, fg, 2);
  di = set(di, bg, 2);
  di = di + step - 5 * 4;

  // Line 6.
  set(di, bg, 4);
}

}  // namespace horus